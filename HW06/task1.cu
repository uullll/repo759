#include "hip/hip_runtime.h"
#include<cstdio>
#include "matmul.cuh"
#include <iostream>
#include <random>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>
using namespace std;

__global__ void task1kernel(float * array, int n){
    int idx=threadIdx.x+blockIdx.x*blockDim.x;
    if (idx<n){
        array[idx]=idx;
    }
}
float* Ramdon_Value(int Min,int Max,int n){
    float *dA=new float[n];
    random_device entropy_source;
    mt19937_64 generator(entropy_source());
    uniform_real_distribution<float> dist(Min, Max);

    for (int i = 0; i < n; ++i) {
        dA[i] = dist(generator);  
    }
    return dA;
}
int main(int argc, char *argv[]){
    if(argc!=3){
        cerr << "Usage: " << argv[0] << " <number_of_size><threads per block>" << endl;
        return 1;
    }
    int n=stoi(argv[1]);
    unsigned int threads_per_block=stoi(argv[2]);
    size_t size = n *n *sizeof(float);
    float *ha=Ramdon_Value(-1.0,1.0,n*n);
    float *hb=Ramdon_Value(-1.0,1.0,n*n);
    float *hc=new float[n*n];

    float *da, *db, *dc;
    hipMalloc(&da,size);
    hipMalloc(&db,size);
    hipMalloc(&dc,size);
    hipMemcpy(da,ha,size,hipMemcpyHostToDevice);
    hipMemcpy(db,hb,size,hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    matmul(da,db,dc,n,threads_per_block);
    hipMemcpy(hc,dc,size,hipMemcpyDeviceToHost);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    cout<<hc[n*n-1]<<endl;
    cout<< milliseconds <<endl;
    
    hipFree(da);
    hipFree(db);
    hipFree(dc);
    delete[] ha;
    delete[] hb;
    delete[] hc;

    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;
}