#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>
#include "matmul.cuh"
#include <random>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>
#include <chrono>
using namespace std;
template <typename T>

__global__ void task1kernel(float * array, int n){
    int idx=threadIdx.x+blockIdx.x*blockDim.x;
    if (idx<n){
        array[idx]=idx;
    }
}
template <typename T>
T *CreateMatrix(int n){
    T *matrix = new T[n];
    random_device entropy_source;
    mt19937_64 generator(entropy_source());
    uniform_real_distribution<T> dist(-10, 10);

    for (int i = 0; i < n; ++i) {
        matrix[i] = dist(generator);  
    }
    return matrix;
}
int *CreateMatrix_1(int n){
    int *matrix = new int[n];
    random_device entropy_source;
    mt19937_64 generator(entropy_source());
    uniform_int_distribution<int> dist(-10, 10);

    for (int i = 0; i < n; ++i) {
        matrix[i] = dist(generator);  
    }
    return matrix;
}
template <typename T>
T *ConvertMatrix(int *matrix, int n){
    T *new_matrix = new T[n];
    for (int i = 0; i < n; ++i) {
        new_matrix[i] = matrix[i];  
    }
    return new_matrix;
}
int main(int argc, char *argv[]){
    if(argc!=3){
        cerr << "Usage: " << argv[0] << " <number_of_size><block_dim>" << endl;
        return 1;
    }
    unsigned int n=atoi(argv[1]);
    unsigned int block_dim=atoi(argv[2]);
    size_t size = n *n *sizeof(float);
    
    int *ha_1=CreateMatrix_1(n*n);
    int *hb_1=CreateMatrix_1(n*n);
    int *hc_1=new int[n*n];

    float *ha_2=CreateMatrix<float>(n*n);
    float *hb_2=CreateMatrix<float>(n*n);
    float *hc_2=new float[n*n];

    double *ha_3=CreateMatrix<double>(n*n);
    double *hb_3=CreateMatrix<double>(n*n);
    double *hc_3=new double[n*n];
    
    int *da_1, *db_1, *dc_1;
    float *da_2, *db_2, *dc_2;
    double *da_3, *db_3, *dc_3;
    hipMalloc(&da_1,size);
    hipMalloc(&db_1,size);
    hipMalloc(&dc_1,size);
    hipMalloc(&da_2,size);
    hipMalloc(&db_2,size);
    hipMalloc(&dc_2,size);
    hipMalloc(&da_3,size);
    hipMalloc(&db_3,size);
    hipMalloc(&dc_3,size);
    hipMemcpy(da_1,ha_1,size,hipMemcpyHostToDevice);
    hipMemcpy(db_1,hb_1,size,hipMemcpyHostToDevice);
    hipMemcpy(da_2,ha_2,size,hipMemcpyHostToDevice);
    hipMemcpy(db_2,hb_2,size,hipMemcpyHostToDevice);
    hipMemcpy(da_3,ha_3,size,hipMemcpyHostToDevice);
    hipMemcpy(db_3,hb_3,size,hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    


    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start, 0);
    matmul_1(da_1,db_1,dc_1,n,block_dim);
    hipMemcpy(hc_1,dc_1,size,hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    
    cout<<hc_1[0]<<endl<<hc_1[n*n-1]<<endl;
    cout<< milliseconds <<endl;


    
    hipEventRecord(start, 0);
    matmul_2(da_2,db_2,dc_2,n,block_dim);
    hipMemcpy(hc_2,dc_2,size,hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout<<hc_2[0]<<endl<<hc_2[n*n-1]<<endl;
    cout<< milliseconds <<endl;

    
    hipEventRecord(start, 0);
    matmul_3(da_3,db_3,dc_3,n,block_dim);
    hipMemcpy(hc_3,dc_3,size,hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout<<hc_3[0]<<endl<<hc_3[n*n-1]<<endl;
    cout<< milliseconds <<endl;


    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(da_1);
    hipFree(db_1);
    hipFree(dc_1);
    hipFree(da_2);
    hipFree(db_2);
    hipFree(dc_2);
    hipFree(da_3);
    hipFree(db_3);
    hipFree(dc_3);
    delete[] ha_1;
    delete[] hb_1;
    delete[] hc_1;
    delete[] ha_2;
    delete[] hb_2;
    delete[] hc_2;
    delete[] ha_3;
    delete[] hb_3;
    delete[] hc_3;
    return 0;      
}