#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include "vscale.cuh"
using namespace std;
float* Ramdon_Value(int Min,int Max,int n){
    float *dA=new float[n];
    random_device entropy_source;
    mt19937_64 generator(entropy_source());
    uniform_real_distribution<float> dist(Min, Max);

    for (int i = 0; i < n; ++i) {
        dA[i] = dist(generator);  
    }
    return dA;
}
int main(int argc, char** argv){

    if (argc < 2) {
        cerr << "Usage: ./task3 <n>" << endl;
        return 1;
    }

    int N = stoi(argv[1]);
    int thread_per_clock = (argc > 2) ? stoi(argv[2]) : 512;
    float *a=Ramdon_Value(-10.0,10.0,N);
    float *b=Ramdon_Value(0.0,1.0,N);
    

    float *ga,*gb;
    hipMalloc(&ga,N*sizeof(float));
    hipMalloc(&gb,N*sizeof(float));
    

    hipMemcpy(a,ga,N*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(b,gb,N*sizeof(float),hipMemcpyHostToDevice);

    

    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    
    int num_blocks=(N+thread_per_clock-1)/thread_per_clock;

    hipEventRecord(start);


    vscale<<<num_blocks,thread_per_clock>>>(ga,gb,N);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float milliseconds=0;
    hipEventElapsedTime(&milliseconds,start,stop);
    hipMemcpy(b,gb,N*sizeof(float),hipMemcpyDeviceToHost);
    cout << "Kernel execution time: " << milliseconds << " ms" << endl;
    cout << "First element of result: " << b[0] << endl;
    cout << "Last element of result: " << b[N - 1] << endl;

    delete[] a;
    delete[] b;
    
    hipFree(ga);
    hipFree(gb);
    

    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;
}